//nvcc k01.cu -o k01 -gencode arch=compute_89,code=compute_89 --threads 0 --std=c++11 -lcufft
#include <hip/hip_runtime.h>


#include <stdio.h>
__global__ void kernel(){
    long blockId = blockIdx.z  *  gridDim.x*gridDim.y
                 + blockIdx.y  *  gridDim.x
                 + blockIdx.x;
    long threadsPerBlock = blockDim.x*blockDim.y*blockDim.z;
    long threadId= threadIdx.z  *  blockDim.x*blockDim.y
                 + threadIdx.y  *  blockDim.x
                 + threadIdx.x
                 + blockId * threadsPerBlock;
    printf("ThreadIdx.x,y,z=%d,%d,%d,%d,%d,%d,%ld\n",
            blockIdx.z,blockIdx.y,blockIdx.x,
            threadIdx.z,threadIdx.y,threadIdx.x,threadId);
}

int main(){
    dim3 threadABlock(2,2,2);
    dim3 blockAGrid(2,2,2);
    kernel<<<blockAGrid,threadABlock>>>();
    hipDeviceSynchronize();
    return 0;
}
