//nvcc k01.cu -o k01
#include <hip/hip_runtime.h>


#include <stdio.h>
__global__ void kernel(){
    long blockId = blockIdx.z  *  gridDim.x*gridDim.y
                 + blockIdx.y  *  gridDim.x
                 + blockIdx.x;
    long threadsPerBlock = blockDim.x*blockDim.y*blockDim.z;
    long threadId= threadIdx.z  *  blockDim.x*blockDim.y
                 + threadIdx.y  *  blockDim.x
                 + threadIdx.x
                 + blockId * threadsPerBlock;
    printf("ThreadIdx.x,y,z=%d,%d,%d,%d,%d,%d,%ld\n",
            threadIdx.x,threadIdx.y,threadIdx.z,
            blockIdx.x,blockIdx.y,blockIdx.z,threadId);
}

int main(){
    dim3 threadABlock(3);
    dim3 blockAGrid(4);
    kernel<<<blockAGrid,threadABlock>>>();
    hipDeviceSynchronize();
    return 0;
}
